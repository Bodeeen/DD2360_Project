#include "hip/hip_runtime.h"
#include "planet.hpp"
#include <iostream>

Planet::Planet(glm::vec4 origin, glm::vec3 lin_vel, glm::vec3 ang_vel, int num_Fe_particles, int num_Si_particles)
{


  setOrigin(origin);
  //setOrigin(glm::vec3(0,0,0));

  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::mt19937 generator(seed);
  std::uniform_real_distribution<double> uniform01(0.0, 1.0);
  std::uniform_real_distribution<double> innerShell(0.0, RInnershell);
  std::uniform_real_distribution<double> outerShell(RInnershell, RImpactor);

  std::cout << num_Si_particles << std::endl;
  std::cout << num_Fe_particles << std::endl;

  for(int i=0; i<num_Si_particles; i++) {
    double rOuter = outerShell(generator);
    double theta = 2 * M_PI * uniform01(generator);
    double phi = acos(1 - 2 * uniform01(generator));
    double x = rOuter * sin(phi) * cos(theta);
    double y = rOuter * sin(phi) * sin(theta);
    double z = rOuter * cos(phi);

    addSilicateParticle(x, y, z);
  }

  for(int i=0; i<num_Fe_particles; i++) {
    double rInner = innerShell(generator);
    double theta = 2 * M_PI * uniform01(generator);
    double phi = acos(1 - 2 * uniform01(generator));
    double x = rInner * sin(phi) * cos(theta);
    double y = rInner * sin(phi) * sin(theta);
    double z = rInner * cos(phi);

    addIronParticle(x, y, z);
  }

  setLinearVelocity(lin_vel, ang_vel);
}

void Planet::setLinearVelocity(glm::vec3 linearVelocity, glm::vec3 angularVelocity)
{
  for(auto &particle : silicateParticles) {
    glm::vec4 arg2 = particle.position - origin;
    glm::vec3 arg = glm::vec3(arg2.x, arg2.y, arg2.z);
    particle.velocity = glm::vec4(glm::cross(angularVelocity / 3600.0f, arg) + linearVelocity, 1.0f);
  }

  for(auto &particle : ironParticles) {
    glm::vec4 arg2 = particle.position - origin;
    glm::vec3 arg = glm::vec3(arg2.x, arg2.y, arg2.z);
    particle.velocity = glm::vec4(glm::cross(angularVelocity / 3600.0f, arg) + linearVelocity, 1.0f);
  }
}

void Planet::setOrigin(glm::vec4 origin)
{
  this->origin = origin;
}

void Planet::setOrigin(double x, double y, double z)
{
  origin.x = x; origin.y = y; origin.z = z; origin.w = 1.0;
}

void Planet::addSilicateParticle(double x_pos, double y_pos, double z_pos)
{
  silicateParticles.push_back(Particle{glm::vec4(x_pos + origin[0], y_pos + origin[1], z_pos + origin[2], 1.0f), glm::vec4(0,0,0,0), 0});
}

void Planet::addIronParticle(double x, double y, double z)
{
  ironParticles.push_back(Particle{glm::vec4(x + origin[0], y + origin[1], z + origin[2], 1.0f), glm::vec4(0,0,0,0), 1});
}

std::vector<Particle> Planet::getIronParticles() { return ironParticles; }

std::vector<Particle> Planet::getSilicateParticles() { return silicateParticles; }

float *Planet::getSilicateColor() { return colorSilicate; }

float *Planet::getIronColor() { return colorIron; }

int Planet::getSilicateCount() { return silicateParticles.size(); }

int Planet::getIronCount() { return ironParticles.size(); }
